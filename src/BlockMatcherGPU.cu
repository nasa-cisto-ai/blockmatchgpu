#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <vector>
#include <nvToolsExt.h>
#include <nvToolsExtCuda.h>
#include "timer.h"
#include "BlockMatcherGPU.h"
#include <iostream>
#include <cmath>

timer t0;

#define NVTX_START(name) nvtxRangePushA(name)
#define NVTX_STOP() nvtxRangePop()

using namespace std;


BlockMatcherGPU::BlockMatcherGPU(int rows, int cols, int block_size, int search_range) {
    r = rows;
    c = cols;
    this->block_size = block_size;
    half_block_size = block_size / 2;
    this->search_range = search_range;
    disparity_map.resize(rows * cols, 0.0);
}

double BlockMatcherGPU::compute_sos(
            const std::vector<double>& kernelCutLeft,
            const std::vector<double>& kernelCutRight
) {

    // return if value is over the max
    if (kernelCutLeft.size() != kernelCutRight.size()) {
        return std::numeric_limits<double>::max();
    }

    double sum_of_squares = 0.0;

    // Iterate over the pixels in the cutouts and compute the Sum of Squares.
    for (size_t i = 0; i < kernelCutLeft.size(); i++) {
        double diff = kernelCutLeft[i] - kernelCutRight[i];
        sum_of_squares += diff * diff;
    }

    return sum_of_squares;
}

__global__ void gpu_sum_of_squares(double* kernel_cut_left, double* kernel_cut_right, int kernel_size, float*  global_sum_of_squares) 
{

    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;

    double sum_of_squares = 0.0;

    // Iterate over the pixels in the cutouts and compute the Sum of Squares.
    for (size_t i = thread_id; i < kernel_size; i += nthreads) {
        double diff = kernel_cut_left[i] - kernel_cut_right[i];
        sum_of_squares += diff * diff;
    }

    // reduction process
    atomicAdd(global_sum_of_squares, (float)sum_of_squares);

}

double BlockMatcherGPU::compute_sos_gpu(
            const std::vector<double>& kernelCutLeft,
            const std::vector<double>& kernelCutRight
) {

    // return if value is over the max
    if (kernelCutLeft.size() != kernelCutRight.size()) {
        return std::numeric_limits<double>::max();
    }

    // defaults from Lars, set via function once we get the GPU kernel working    
    int rows = 1000;
    int cols = 1000;
    dim3 threads = {1024};
    dim3 blocks;

    NVTX_START("Initializing Cuda inside compute Disparity");
    hipFree(0);
    int deviceID;
    hipDeviceProp_t prop;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&prop, deviceID); 
    blocks = {(unsigned)prop.multiProcessorCount};
    NVTX_STOP();

    // overall variables to fill in
    double sum_of_squares = 0.0;
    float* global_sum_of_squares;

    NVTX_START("Allocating and copying GPU memory");
    //cout << "Allocating and copying GPU memory\n";
    
    double* kernelCutLeft_device;
    double* kernelCutRight_device;
    float* global_sum_of_squares_device;

    //hipMallocManaged(&p, kernelCutRight.size()*sizeof(int));
    hipMallocManaged(&kernelCutLeft_device, kernelCutRight.size()*sizeof(double));
    hipMallocManaged(&kernelCutRight_device, kernelCutRight.size()*sizeof(double));
    hipMallocManaged(&global_sum_of_squares_device, sizeof(float));

    hipError_t err = hipGetLastError();
    if (err) {
        printf("GPU Error: %s\n", hipGetErrorString(err));
    }

    //cout << "Done with hipMallocManaged\n";
    //cout << "Size " << kernelCutRight.size()<< "\n";

    // add the values to the GPU - kernelCutLeft, kernelCutRight
    for (int j = 0; j < kernelCutRight.size(); j++) {
        //cout << j << "\n";
        kernelCutLeft_device[j] = kernelCutLeft[j];
        kernelCutRight_device[j] = kernelCutRight[j];
    }
    NVTX_STOP();

    //cout << "Done with hipMallocManaged assignments\n";

    // initialize global_sum_of_squares
    *global_sum_of_squares_device = 0.0;

    //cout << "Starting GPU processing\n";

    NVTX_START("GPU Exec");
    t0.tick();
    gpu_sum_of_squares<<<blocks, threads>>>(kernelCutLeft_device, kernelCutRight_device, kernelCutRight.size(), global_sum_of_squares_device);
    hipDeviceSynchronize();

    err = hipGetLastError();
    if (err) {
        printf("GPU Error: %s\n", hipGetErrorString(err));
    }

    double dt = t0.tock();
    NVTX_STOP();

    //cout << "Finished GPU processing, returning global sum of squares\n";

    //hipFree(kernelCutLeft_device);
    //hipFree(kernelCutRight_device);
    //hipFree(global_sum_of_squares_device);

    return *global_sum_of_squares_device;
}

double BlockMatcherGPU::compute_box_sum(const std::vector<double>& kernelCutLeft,
                        const std::vector<double>& kernelCutRight) {
    double box_sum = 0.0;

    // Iterate over the pixels in the cutouts and compute the box sum
    for (size_t i = 0; i < kernelCutLeft.size(); i++) {
        double diff = kernelCutLeft[i] - kernelCutRight[i];
        box_sum += std::abs(diff); // Absolute difference for the box sum
    }

    return box_sum;
}

void BlockMatcherGPU::compute_disparity(const std::vector<double>& left_image, const std::vector<double>& right_image) {

    int max_displacement = search_range;

    for (int i = half_block_size; i < r - half_block_size; i++) {
        for (int j = half_block_size; j < c - half_block_size; j++) {
            
            // Take a cutout centered on each pixel in the left image
            std::vector<double> kernelCutLeft(block_size * block_size, 0.0);
            for (int y = -half_block_size; y <= half_block_size; y++) {
                for (int x = -half_block_size; x <= half_block_size; x++) {
                    kernelCutLeft[(y + half_block_size) * block_size + (x + half_block_size)] = left_image[(i + y) * c + (j + x)];
                }
            }

            int min_disparity = 0;
            double min_sos = std::numeric_limits<double>::max();
            // std::cout << i << " ";
            // std::cout << std::endl;

            // Search within the specified search range
            for (int d = 0; d <= max_displacement; d++) {
                // std::cout << "Searching disparity range" << d << " ";
                // std::cout << std::endl;
                // Shift the right image by the current disparity
                std::vector<double> kernelCutRight(block_size * block_size, 0.0);
                for (int y = -half_block_size; y <= half_block_size; y++) {
                    for (int x = -half_block_size; x <= half_block_size; x++) {
                        kernelCutRight[(y + half_block_size) * block_size + (x + half_block_size)] = right_image[(i + y) * c + (j + x - d)];
                    }
                }

                // Compute the Sum of Squares (SOS) between the cutout and the matching cutout
                //double sos = compute_sos(kernelCutLeft, kernelCutRight);
                //cout << "Sum of squares output " << sos << " \n";

                // call the global function here
                //cout << "Calling compute_sos_gpu\n";
                //cout << "Left side kernel " << kernelCutLeft.size() << "\n";
                //cout << "Right side kernel " << kernelCutRight.size() << "\n";

                double sos = compute_sos_gpu(kernelCutLeft, kernelCutRight);

                //cout << "Done with compute_sos_gpu\n";

                // for loop to sum the values

                // Update the disparity if the SOS is smaller
                if (sos < min_sos) {
                    min_sos = sos;
                    min_disparity = d;
                }
            }

            // Store the disparity in the disparity map
            disparity_map[i * c + j] = min_disparity;
        }
    }
}

std::vector<double>& BlockMatcherGPU::getDisparityMap() {
    return disparity_map;
}
